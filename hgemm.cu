#include "hip/hip_runtime.h"
#include <unistd.h>
#include <iostream>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "helper.h"

using namespace std;

int main(int argc, char ** argv){

  int max_m_k_n = 2028;
  int repeats = 10;
  int verbose = 1;

  cout << "\nrunning hipblasHgemm test\n" << endl;
  
  if(verbose) 
    cout << "running with" 
	 << " max_m_k_n: " << max_m_k_n
	 << " repeats: " << repeats
	 << endl;

  hipblasStatus_t stat;
  hipblasHandle_t handle;

  checkCublas(hipblasCreate(&handle));

  if(verbose) cout << "allocating device variables" << endl;
  
  // Allocate 3 arrays on CPU
  
  float *h_A = (float *)malloc(max_m_k_n * max_m_k_n * sizeof(float));
  float *h_B = (float *)malloc(max_m_k_n * max_m_k_n * sizeof(float));
  float *h_C = (float *)malloc(max_m_k_n * max_m_k_n * sizeof(float));
  
  CPU_fill_rand(h_A, max_m_k_n, max_m_k_n);
  CPU_fill_rand(h_B, max_m_k_n, max_m_k_n);
  CPU_fill_rand(h_C, max_m_k_n, max_m_k_n);
    
	__half *d_A, *d_B, *d_C;
  checkCuda(hipMallocManaged(&d_A, max_m_k_n * max_m_k_n * sizeof(__half)));
  checkCuda(hipMallocManaged(&d_B, max_m_k_n * max_m_k_n * sizeof(__half)));
  checkCuda(hipMallocManaged(&d_C, max_m_k_n * max_m_k_n * sizeof(__half)));
  
  for (int i = 0; i < max_m_k_n * max_m_k_n; i++) {
    d_A[i] = approx_float_to_half(h_A[i]);
	  d_B[i] = approx_float_to_half(h_B[i]);
	  d_C[i] = approx_float_to_half(h_C[i]);
  }
  
  int lda, ldb, ldc, m, n, k;
  const __half alf = approx_float_to_half(1.0);
  const __half bet = approx_float_to_half(0.0);
  const __half *alpha = &alf;
  const __half *beta = &bet;

  cout << "begin testing: \n";
  
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  for(int i = 0; i < 1; i++){
    double sum = 0.0;

    for(int rep = 0; rep < repeats; rep++){
      hipEventRecord(start, 0);
    	  m=256;
        n=256;
        k=256;
    	  lda = m;
    	  ldb = k;
    	  ldc = m;

      	stat = hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, d_A, lda, d_B, ldb, beta, d_C, ldc); 

      hipEventRecord(stop,0);
      hipEventSynchronize(stop);

      if(stat != HIPBLAS_STATUS_SUCCESS){
      	cerr << "hipblasSgemmBatched failed" << endl;
      	exit(1);
      }

      assert(!hipGetLastError());
      
      float elapsed;
      hipEventElapsedTime(&elapsed, start, stop);
      elapsed /= 1000.0f;
      sum += elapsed;
    }

    cout << "float16; " << " average: " << sum/repeats << " s "<< endl;
  }

  //Free GPU memory
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  // Free CPU memory
  free(h_A);
  free(h_B);
  free(h_C);
      
  return 0;
}
